#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "kernels.h"


// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C,int nums)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//C[i] = A[i] + B[i];
	if(i<nums){
		//C[i]=A[i]+B[i];
		C[i]=static_cast<float>(i);
	}
}

void printerr(hipError_t err){
	if (err != hipSuccess)
	{
		fprintf(stderr, "error code %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void VecAddWrapper(float* A, float* B, float* C,int Count){
	hipError_t err = hipSuccess;
	size_t size=Count*sizeof(float);
	float *d_A = NULL;
	err = hipMalloc(&d_A, size);
	printerr(err);
	float *d_B = NULL;
	err = hipMalloc(&d_B, size);
	printerr(err);
	float *d_C = NULL;
	err = hipMalloc(&d_C, size);
	printerr(err);
	err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	printerr(err);
	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	printerr(err);
	dim3 tpB(128,1);
	VecAdd<<<1, tpB>>>(d_A, d_B, d_C, Count);
	printerr(err);
	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printerr(err);
	err = hipFree(d_A);
	printerr(err);
	err = hipFree(d_B);
	printerr(err);
	err = hipFree(d_C);
	printerr(err);
}
